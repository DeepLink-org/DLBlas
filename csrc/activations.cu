#include "hip/hip_runtime.h"
// modify from: https://github.com/vllm-project/vllm
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>

#include "cuda_compat.h"
#include "dispatch_utils.h"


namespace dlblas {

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          bool act_first>
__device__ __forceinline__ scalar_t compute(const scalar_t& x,
                                            const scalar_t& y) {
  return act_first ? ACT_FN(x) * y : x * ACT_FN(y);
}
// Activation and gating kernel template.

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          bool act_first>
__global__ void act_and_mul_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = DLBLAS_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = DLBLAS_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = compute<scalar_t, ACT_FN, act_first>(x, y);
  }
}

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

}  // namespace dlblas

// Launch activation and gating kernel.
// Use ACT_FIRST (bool) indicating whether to apply the activation function
// first.
#define LAUNCH_ACTIVATION_GATE_KERNEL(KERNEL, ACT_FIRST)                 \
  int d = input.size(-1) / 2;                                            \
  int64_t num_tokens = input.numel() / input.size(-1);                   \
  dim3 grid(num_tokens);                                                 \
  dim3 block(std::min(d, 1024));                                         \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));      \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();          \
  VLLM_DISPATCH_FLOATING_TYPES(                                          \
      input.scalar_type(), "act_and_mul_kernel", [&] {                   \
        dlblas::act_and_mul_kernel<scalar_t, KERNEL<scalar_t>, ACT_FIRST>  \
            <<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(),       \
                                         input.data_ptr<scalar_t>(), d); \
      });


void silu_and_mul(torch::Tensor& out,    // [..., d]
                  torch::Tensor& input)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL(dlblas::silu_kernel, true);
}
